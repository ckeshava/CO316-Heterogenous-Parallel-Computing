
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>

__global__ void add(float *A, float *B, float *C, int N)
{
    int thread_index = blockDim.x * blockIdx.x + threadIdx.x;

    if (thread_index < N)
        C[thread_index] = A[thread_index] + B[thread_index];
}

void generate_floats(float *A, int N)
{
    for (int i = 0; i < N; ++i)
        A[i] = sin(i) + cos(i);
}

int main()
{
    printf("\n\nProgram to perform Vector Addition in CUDA\n\n");
    int N = 2048; // Number of elements in the array

    float *A, *B, *C;
    float host_A[N], host_B[N], host_C[N];

    // generate random floating numbers for input
    printf("\nGenerating %d floating-point numbers for the input arrays....\n", N);
    generate_floats(host_A, N);
    generate_floats(host_B, N);

    printf("\nAllocating memory on the GPU...\n\n");
    // allocate space on device
    hipMalloc((void **)&A, N * sizeof(float));
    hipMalloc((void **)&B, N * sizeof(float));
    hipMalloc((void **)&C, N * sizeof(float));

    // memory transfer from host to device
    printf("\nTransferring data from host to device for computations...\n\n");

    hipMemcpy(A, host_A, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(B, host_B, N * sizeof(float), hipMemcpyHostToDevice);

    // dimensions of thread block + kernel launch
    int blockDim = 1024;

    int gridDim = ceil((float)(N) / 1024);

    printf("\n\nCalling the kernel with %d Blocks and %d threads in each block\n", gridDim, blockDim);

    add<<<gridDim, blockDim>>>(A, B, C, N);

    // copy back to host
    printf("\n\nCalculation completed on the GPU. Fetching the answer back from the GPU's global memory\n");
    hipMemcpy(host_C, C, N * sizeof(float), hipMemcpyDeviceToHost);

    // free the malloc'ed memory
    printf("\n\nFree'ing the malloc'ed memory on the GPU\n");
    hipFree(A);
    hipFree(B);
    hipFree(C);

    return 0;
}